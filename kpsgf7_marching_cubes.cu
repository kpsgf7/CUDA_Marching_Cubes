#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include <math.h>
#include <time.h>

// includes, kernels
#include <hip/hip_runtime.h>
#include "kpsgf7_marching_cubes_kernel.cu"

// includes, project
#include <helper_functions.h>  // helper for shared that are common to CUDA SDK samples
#include <hip/hip_runtime_api.h>       // helper for checking cuda initialization and error checking
#include "kpsgf7_marching_cubes_helpers.cu"




int main(int argc, char **argv){

	if (argc != 6){
		printf("ERROR: IMPROPER PROGRAM USAGE\nCorrect usage: ./kpsgf7_marching_cubes <threshold value> <input directory> <number of input slices> <input image step> <output file>\nTerminating Program...\n");
		return -1;
	}
	hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    int dev = 0;

    // This will pick the best possible CUDA capable device
    dev = findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if (version < 0x20)
    {
        printf(" requires a minimum CUDA compute 2.0 capability\n");
        exit(EXIT_SUCCESS);
    }


    // Load one image to determine the width and height of the passed images
    unsigned char *h_img0c = NULL;

    unsigned int w, h;
    std::string directory(argv[2]);

    std::string filepath = directory + "/0.pgm";

    const char *fname = filepath.c_str();

    if(!sdkLoadPGM(fname, &h_img0c, &w, &h)){
    	fprintf(stderr, "Failed to load <%s>\n", fname);
    	return -1;
    }

    printf("Loaded <%s> as image 0\n", fname);

    // using that width and height, allocate host memory
    unsigned int input_slice_n = atoi(argv[3]);
    unsigned int numData = w*h;
    unsigned int memSize = sizeof(unsigned int) * numData * input_slice_n;

    unsigned int *host_images = (unsigned int*)malloc(memSize);
   
    if(host_images == NULL){
    	printf("Failed to allocate. Exiting.\n");
    	return -1;
    }
 
    // cast and copy in the first image
    for (int i=0; i<numData; i++){
    	host_images[i] = (unsigned int)h_img0c[i];
    }

    int step = atoi(argv[4]);

    // load the remainder of the images into the host memory
    for (int img_idx = 1; img_idx<input_slice_n; img_idx++){

        filepath = directory + "/" + std::to_string(img_idx*step)  + ".pgm";

        fname = filepath.c_str();

        if(!sdkLoadPGM(fname, &h_img0c, &w, &h)){
            fprintf(stderr, "Failed to load <%s>\n", fname);
            return -1;
        }   

        for (int i=0; i<numData; i++){
        host_images[i+img_idx * numData] = (unsigned int)h_img0c[i];
        }
    }


    printf("Allocated Host Memory\n");

    //set up timing events
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start,NULL));

    // allocate device memory 
    unsigned int *device_output, *device_images;

    checkCudaErrors(hipMalloc((void **) &device_output, memSize));
    checkCudaErrors(hipMalloc((void **) &device_images, memSize));

    //copy host mem to device memory
    checkCudaErrors(hipMemcpy(device_images,  host_images, memSize, hipMemcpyHostToDevice));

    unsigned int thresh = atoi(argv[1]);
    std::cout << "Launching thresholding kernel\n";

    dim3 grid_size_thresh(32,32,input_slice_n);
    dim3 block_size_thresh(16,16,1);
    thresholding_filter_kernel<<<grid_size_thresh,block_size_thresh>>>(device_images, device_output ,thresh);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipPeekAtLastError());

    //uncomment to see intermediate images

    // checkCudaErrors(hipMemcpy(host_images,  device_output, memSize, hipMemcpyDeviceToHost));
    // int out=0;
    // for(out=0; out<input_slice_n; out++){

    //     unsigned char *hold = (unsigned char*)malloc(sizeof(unsigned char) * numData);

    //     for (int i=0; i<numData; i++){
    //         hold[i] = (unsigned char)host_images[i + out * numData];
    //     }

    //     // save output
    //     std::string fname_out = "output_scan/" + std::to_string(out) + ".pgm";
    //     printf("Saving file at %s\n", fname_out.c_str());
    //     sdkSavePGM(fname_out.c_str(), hold, w, h);
    // }


    // clean up after the thresholding and set up for marching cubes
    hipFree(device_images);
    //free(host_images);
    free(h_img0c);
    int *host_lookup_one = get_lookup_one();
    int *host_lookup_two = get_lookup_two();



    std::cout << "\nAllocating Cubes memory\n";
    int *device_lookup_one;
    int *device_lookup_two;

    float *device_triangles;
    int max_triangle_points = (5 * 3 *3 * numData * input_slice_n);
    float *host_triangles = (float *)malloc(sizeof(float) *max_triangle_points);
    if (host_triangles == NULL){
    	std::cout << "Allocating failed\nTerminating\n";
    	return -1;
    }

    std::cout << max_triangle_points * sizeof(float) << " bytes allocated for triangles\n";

    checkCudaErrors(hipMalloc((void **) &device_triangles, sizeof(float) * max_triangle_points));
    checkCudaErrors(hipMalloc((void **) &device_lookup_one, sizeof(int) * 256));
    checkCudaErrors(hipMalloc((void **) &device_lookup_two, sizeof(int) * 256 * 16));

    //checkCudaErrors(hipMemcpy(device_lookup_one,  host_lookup_one, sizeof(int)*256, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_lookup_two,  host_lookup_two, sizeof(int) * 256 * 16, hipMemcpyHostToDevice));
    std::cout << "Launching cubes kernel\n";
   

    dim3 grid_size(64,64,input_slice_n/4); //512,512
    dim3 block_size(8,8,8);
    marching_cubes_filter<<<grid_size, block_size>>>(device_output, device_lookup_one, device_lookup_two, device_triangles,w,h,input_slice_n,step);
    hipDeviceSynchronize();

    std::cout << hipGetErrorString(hipPeekAtLastError());

    //Copy result from device to host for verification
    checkCudaErrors(hipMemcpy(host_triangles, device_triangles, sizeof(float)*max_triangle_points, hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(stop,NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    std::cout << "\nGPU processing time : " << msecTotal << " (ms)\n";

   
    // // write a ply file
    write_ply(host_triangles,max_triangle_points, argv[5]);
   
    hipFree(device_triangles);
    hipFree(device_output);
    

    std::cout << "Comparing GPU time to CPU time\n";
    clock_t cpu_start = clock();
    compute_cpu_marching_cubes(host_images, thresh, w, h, input_slice_n, host_lookup_one, host_lookup_two, host_triangles);
    clock_t cpu_end = clock();
    std::cout << "CPU processing time: " << (((double)(cpu_end - cpu_start)/ CLOCKS_PER_SEC)*1000) << " ms\n";

    free(host_images);
    free(host_triangles);


    return 0;

}

